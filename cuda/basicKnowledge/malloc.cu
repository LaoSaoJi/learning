#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int size = 10 * sizeof(int);
    int* d_data;
    hipError_t hipError_t = hipMalloc((void**)&d_data, size);
    if (hipError_t != hipSuccess) {
        printf("hipMalloc failed: %s\n", hipGetErrorString(hipError_t));
        return 1;
    }
    printf("hipMalloc succeeded, d_data\n");
    hipFree(d_data);
}