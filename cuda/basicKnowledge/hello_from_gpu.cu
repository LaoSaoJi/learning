// how to use gpu, how to write kernel function.
// what is block and grid.
// what is warp and how to get warpSize.

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello() {
    printf("hello world from gpu!\n");
    printf("warpSize: %d", warpSize);
}

int main() {
    dim3 block(1, 2);
    dim3 grid(1, 2);
    hello<<<grid, 1>>>();
    hipDeviceSynchronize();
    return 0;
}